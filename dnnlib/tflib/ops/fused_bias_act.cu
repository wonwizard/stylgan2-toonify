#include "hip/hip_runtime.h"
// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

#define EIGEN_USE_GPU
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS__
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/framework/shape_inference.h"
#include <stdio.h>

using namespace tensorflow;
using namespace tensorflow::shape_inference;

#define OP_CHECK_CUDA_ERROR(CTX, CUDA_CALL) do { hipError_t err = CUDA_CALL; OP_REQUIRES(CTX, err == hipSuccess, errors::Internal(hipGetErrorName(err))); } while (false)

//------------------------------------------------------------------------
// CUDA kernel.

template <class T>
struct FusedBiasActKernelParams
{
    const T*    x;      // [sizeX]
    const T*    b;      // [sizeB] or NULL
    const T*    ref;    // [sizeX] or NULL
    T*          y;      // [sizeX]

    int         grad;
    int         axis;
    int         act;
    float       alpha;
    float       gain;

    int         sizeX;
    int         sizeB;
    int         stepB;
    int         loopX;
};

template <class T>
static __global__ void FusedBiasActKernel(const FusedBiasActKernelParams<T> p)
{
    const float expRange        = 80.0f;
    const float halfExpRange    = 40.0f;
    const float seluScale       = 1.0507009873554804934193349852946f;
    const float seluAlpha       = 1.6732632423543772848170429916717f;

    // Loop over elements.
    int xi = blockIdx.x * p.loopX * blockDim.x + threadIdx.x;
    for (int loopIdx = 0; loopIdx < p.loopX && xi < p.sizeX; loopIdx++, xi += blockDim.x)
    {
        // Load and apply bias.
        float x = (float)p.x[xi];
        if (p.b)
            x += (float)p.b[(xi / p.stepB) % p.sizeB];
        float ref = (p.ref) ? (float)p.ref[xi] : 0.0f;
        if (p.gain != 0.0f & p.act != 9)
            ref /= p.gain;

        // Evaluate activation func.
        float y;
        switch (p.act * 10 + p.grad)
        {
            // linear
            default:
            case 10: y = x; break;
            case 11: y = x; break;
            case 12: y = 0.0f; break;

            // relu
            case 20: y = (x > 0.0f) ? x : 0.0f; break;
            case 21: y = (ref > 0.0f) ? x : 0.0f; break;
            case 22: y = 0.0f; break;

            // lrelu
            case 30: y = (x > 0.0f) ? x : x * p.alpha; break;
            case 31: y = (ref > 0.0f) ? x : x * p.alpha; break;
            case 32: y = 0.0f; break;

            // tanh
            case 40: { float c = expf(x); float d = 1.0f / c; y = (x < -expRange) ? -1.0f : (x > expRange) ? 1.0f : (c - d) / (c + d); } break;
            case 41: y = x * (1.0f - ref * ref); break;
            case 42: y = x * (1.0f - ref * ref) * (-2.0f * ref); break;

            // sigmoid
            case 50: y = (x < -expRange) ? 0.0f : 1.0f / (expf(-x) + 1.0f); break;
            case 51: y = x * ref * (1.0f - ref); break;
            case 52: y = x * ref * (1.0f - ref) * (1.0f - 2.0f * ref); break;

            // elu
            case 60: y = (x >= 0.0f) ? x : expf(x) - 1.0f; break;
            case 61: y = (ref >= 0.0f) ? x : x * (ref + 1.0f); break;
            case 62: y = (ref >= 0.0f) ? 0.0f : x * (ref + 1.0f); break;

            // selu
            case 70: y = (x >= 0.0f) ? seluScale * x : (seluScale * seluAlpha) * (expf(x) - 1.0f); break;
            case 71: y = (ref >= 0.0f) ? x * seluScale : x * (ref + seluScale * seluAlpha); break;
            case 72: y = (ref >= 0.0f) ? 0.0f : x * (ref + seluScale * seluAlpha); break;

            // softplus
            case 80: y = (x > expRange) ? x : logf(expf(x) + 1.0f); break;
            case 81: y = x * (1.0f - expf(-ref)); break;
            case 82: { float c = expf(-ref); y = x * c * (1.0f - c); } break;

            // swish
            case 90: y = (x < -expRange) ? 0.0f : x / (expf(-x) + 1.0f); break;
            case 91: { float c = expf(ref); float d = c + 1.0f; y = (ref > halfExpRange) ? x : x * c * (ref + d) / (d * d); } break;
            case 92: { float c = expf(ref); float d = c + 1.0f; y = (ref > halfExpRange) ? 0.0f : x * c * (ref * (2.0f - d) + 2.0f * d) / (d * d * d); } break;
        }

        // Apply gain and store.
        p.y[xi] = (T)(y * p.gain);
    }
}

//------------------------------------------------------------------------
// TensorFlow op.

template <class T>
struct FusedBiasActOp : public OpKernel
{
    FusedBiasActKernelParams<T> m_attribs;

    FusedBiasActOp(OpKernelConstruction* ctx) : OpKernel(ctx)
    {
        memset(&m_attribs, 0, sizeof(m_attribs));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("grad", &m_attribs.grad));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("axis", &m_attribs.axis));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("act", &m_attribs.act));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("alpha", &m_attribs.alpha));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("gain", &m_attribs.gain));
        OP_REQUIRES(ctx, m_attribs.grad >= 0, errors::InvalidArgument("grad must be non-negative"));
        OP_REQUIRES(ctx, m_attribs.axis >= 0, errors::InvalidArgument("axis must be non-negative"));
        OP_REQUIRES(ctx, m_attribs.act >= 0, errors::InvalidArgument("act must be non-negative"));
    }

    void Compute(OpKernelContext* ctx)
    {
        FusedBiasActKernelParams<T> p = m_attribs;
        hipStream_t stream = ctx->eigen_device<Eigen::GpuDevice>().stream();

        const Tensor& x     = ctx->input(0); // [...]
        const Tensor& b     = ctx->input(1); // [sizeB] or [0]
        const Tensor& ref   = ctx->input(2); // x.shape or [0]
        p.x = x.flat<T>().data();
        p.b = (b.NumElements()) ? b.flat<T>().data() : NULL;
        p.ref = (ref.NumElements()) ? ref.flat<T>().data() : NULL;
        OP_REQUIRES(ctx, b.NumElements() == 0 || m_attribs.axis < x.dims(), errors::InvalidArgument("axis out of bounds"));
        OP_REQUIRES(ctx, b.dims() == 1, errors::InvalidArgument("b must have rank 1"));
        OP_REQUIRES(ctx, b.NumElements() == 0 || b.NumElements() == x.dim_size(m_attribs.axis), errors::InvalidArgument("b has wrong number of elements"));
        OP_REQUIRES(ctx, ref.NumElements() == ((p.grad == 0) ? 0 : x.NumElements()), errors::InvalidArgument("ref has wrong number of elements"));
        OP_REQUIRES(ctx, x.NumElements() <= kint32max, errors::InvalidArgument("x is too large"));

        p.sizeX = (int)x.NumElements();
        p.sizeB = (int)b.NumElements();
        p.stepB = 1;
        for (int i = m_attribs.axis + 1; i < x.dims(); i++)
            p.stepB *= (int)x.dim_size(i);

        Tensor* y = NULL; // x.shape
        OP_REQUIRES_OK(ctx, ctx->allocate_output(0, x.shape(), &y));
        p.y = y->flat<T>().data();

        p.loopX = 4;
        int blockSize = 4 * 32;
        int gridSize = (p.sizeX - 1) / (p.loopX * blockSize) + 1;
        void* args[] = {&p};
        OP_CHECK_CUDA_ERROR(ctx, hipLaunchKernel((void*)FusedBiasActKernel<T>, gridSize, blockSize, args, 0, stream));
    }
};

REGISTER_OP("FusedBiasAct")
    .Input      ("x: T")
    .Input      ("b: T")
    .Input      ("ref: T")
    .Output     ("y: T")
    .Attr       ("T: {float, half}")
    .Attr       ("grad: int = 0")
    .Attr       ("axis: int = 1")
    .Attr       ("act: int = 0")
    .Attr       ("alpha: float = 0.0")
    .Attr       ("gain: float = 1.0");
REGISTER_KERNEL_BUILDER(Name("FusedBiasAct").Device(DEVICE_GPU).TypeConstraint<float>("T"), FusedBiasActOp<float>);
REGISTER_KERNEL_BUILDER(Name("FusedBiasAct").Device(DEVICE_GPU).TypeConstraint<Eigen::half>("T"), FusedBiasActOp<Eigen::half>);

//------------------------------------------------------------------------
