// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html


#include <hip/hip_runtime.h>
#include <cstdio>

void checkCudaError(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        exit(1);
    }
}

__global__ void cudaKernel(void)
{
    printf("GPU says hello.\n");
}

int main(void)
{
    printf("CPU says hello.\n");
    checkCudaError(hipLaunchKernel(reinterpret_cast<const void*>((void*)cudaKernel), 1, 1, NULL, 0, NULL));
    checkCudaError(hipDeviceSynchronize());
    return 0;
}
